#include "hip/hip_runtime.h"
#include <stdio.h>

//DEVICE

__device__ int filtrarPixel(int* imgX, int X, int Y, int nLinhas, int nColunas,int TamFiltro)
{
	int i, j;
	int total = 0;
	int pixX, pixY;

	int a = X-(TamFiltro/2);
	int b = Y-(TamFiltro/2);

	int offset;


	for(i = 0; i<TamFiltro; i++)
	{
		for(j= 0; j<TamFiltro; j++)
		{
			pixX = a +i;
			pixY = b +j;

			if(!(pixX < 0 || pixY < 0 || pixX >= nColunas || pixY >= nLinhas))
			{
				offset = pixX*nColunas+pixY;

				total = total + imgX[offset];
			}
		}
	}
	total = total/(TamFiltro*TamFiltro);
	return(total);
}

__global__ void filtroGuassiano(int* imgX, int* imgXF, int nLinhas, int nColunas, int tamFiltro)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	int j = (blockIdx.y * blockDim.y) + threadIdx.y;

	if( i< nColunas && j< nLinhas)
	{
		//Processa!!!
		int offset = j*nColunas+i;
		imgXF[offset] = filtrarPixel(imgX, i, j, nLinhas, nColunas, tamFiltro);
		//imgXF[offset] = imgX[offset]+1;	
		//imgXF[offset] =  returnResult(imgX, offset);
	}
}


//HOST

typedef struct header
{
	char P[3];
	int x;
	int y;
	int max;
}HEADER;

typedef struct pixel{
	int R;
	int G;
	int B;
}PIXEL;

void lerPixel(int* pixR, int* pixG, int* pixB, FILE* img)
{
	fscanf(img, "%d", pixR);
	fscanf(img, "%d", pixG);
	fscanf(img, "%d", pixB);
}

void lerHeader(HEADER *head, FILE* img)
{
	char c;
	fread(head->P, sizeof(char), 2, img);
	head->P[2] = '\0';
	fseek(img, 1, SEEK_CUR);

	fread(&c, sizeof(char), 1, img);
	if(c == '#')
	{
		do{
			fread(&c, sizeof(char), 1, img);
		}while(c != '\n');
	}
	else
	fseek(img, -1, SEEK_CUR);
	fscanf(img, "%d", &head->x);
	fscanf(img, "%d", &head->y);
	fscanf(img, "%d", &head->max);

	//printf("%s\n", head->P);
	//printf("%d %d\n", head->x, head->y);

}

void lerImagem(HEADER* head, FILE* img, int** imgR, int** imgG, int** imgB)
{
	int i, j;

	for(i = 0 ; i < head->x; i++)
	{
		for(j = 0; j < head->y; j++)
		{
			lerPixel(&imgR[i][j], &imgG[i][j], &imgB[i][j], img);
			//printf("%d %d %d  ", imgR[i][j], imgG[i][j], imgB[i][j]);
		}
		//printf("\n");
	}
}

void escreverImagem(int** imgR, int** imgG, int** imgB,  HEADER* head, FILE* imgSaida)
{
	int i, j;
	fseek(imgSaida, 0, SEEK_SET);

	fprintf(imgSaida, "%s\n%d %d\n%d\n", head->P, head->x, head->y, head->max);

	for(i = 0; i < head->x; i++)
	{
		for(j = 0; j < head->y; j++)
		{
			fprintf(imgSaida, "%d %d %d\n", imgR[i][j], imgG[i][j], imgB[i][j]);
		}
	}
}

void escreverImagemVetor(int* imgVR, int* imgVG, int* imgVB,  HEADER* head, FILE* imgSaida)
{
	int i, j;
	int nColunas = head->x;
	int offset;

	fseek(imgSaida, 0, SEEK_SET);

	fprintf(imgSaida, "%s\n%d %d\n%d\n", head->P, head->x, head->y, head->max);

	for(i = 0; i < head->x; i++)
	{
		for(j = 0; j < head->y; j++)
		{
			offset = i*nColunas+j;
			//printf("i = %d j = %d offset = %d\n", i, j, offset);
			fprintf(imgSaida, "%d %d %d\n", imgVR[offset], imgVG[offset], imgVB[offset]);
		}
	}
}

void lerImagemVetor(HEADER* head, FILE* img, int* imgVR, int* imgVG, int* imgVB)
{
	int i, j;
	int nColunas = head->x;
	int offset;

	//printf("nColunas = %d nLinhas = %d\n", head->x, head->y);

	for(i = 0 ; i < head->x; i++)
	{
		for(j = 0; j < head->y; j++)
		{

			offset = i*nColunas+j;
			//printf("i = %d j = %d offset = %d\n", i, j, offset);
			lerPixel(&imgVR[offset], &imgVG[offset], &imgVB[offset], img);
		}
	}
}


int** alocarMatriz(int x, int y)
{
	int i;
	int **mat;

	mat = (int**) malloc (sizeof(int*)*x);
	if(mat == NULL)
	{
		printf("Erro ao alocar matriz\n");
		exit(EXIT_FAILURE);
	}

	for(i = 0; i<x;i++)
	{

		mat[i] = (int*) malloc(sizeof(int)*y);
		if(mat[i] == NULL)
		{
			printf("Erro ao alocar matriz\n");
			exit(EXIT_FAILURE);
		}
	}
	return(mat);
}

void lerImagemGray(HEADER* head, FILE* img, int** imgGray)
{
	int i, j;

	for(i = 0 ; i < head->x; i++)
	{
		for(j = 0; j < head->y; j++)
		{
			fscanf(img, "%d", &imgGray[i][j]);
			//printf("%d %d %d  ", imgR[i][j], imgG[i][j], imgB[i][j]);
		}
		//printf("\n");
	}
}

void escreverImagemGray(int** imgG, HEADER* head, FILE* imgSaida)
{
	int i, j;
	fseek(imgSaida, 0, SEEK_SET);

	fprintf(imgSaida, "%s\n%d %d\n%d\n", head->P, head->x, head->y, head->max);

	for(i = 0; i < head->x; i++)
	{
		for(j = 0; j < head->y; j++)
		{
			fprintf(imgSaida, "%d \n", imgG[i][j]);
		}
	}
}

void lerImagemGrayVetor(HEADER* head, FILE* img, int* imgVGray)
{
	int i, j;
	int nColunas = head->x;
	int offset;

	for(i = 0 ; i < head->x; i++)
	{
		for(j = 0; j < head->y; j++)
		{
			offset = i*nColunas+j;
			fscanf(img, "%d", &imgVGray[offset]);
			//printf("%d %d %d  ", imgR[i][j], imgG[i][j], imgB[i][j]);
		}
		//printf("\n");
	}
}

void escreverImagemGrayVetor(int* imgVG, HEADER* head, FILE* imgSaida)
{
	int i, j;
	int nColunas = head->x;
	int offset;

	fseek(imgSaida, 0, SEEK_SET);

	fprintf(imgSaida, "%s\n%d %d\n%d\n", head->P, head->x, head->y, head->max);

	for(i = 0; i < head->x; i++)
	{
		for(j = 0; j < head->y; j++)
		{
			offset = i*nColunas+j;
			fprintf(imgSaida, "%d \n", imgVG[offset]);
		}
	}
}

void desalocarMatriz(int x, int **matriz)
{
	int i;
	for(i = 0; i<x; i++)
	{
    	free(matriz[i]);
	}
	free(matriz);
}

#define MAXTHREADS 4

int main( int argc, char* argv[] )
{

    int nTX, nTY;

    int n;
    int nColunas;
    int nLinhas;

    hipError_t error;
    HEADER head;

	FILE* img;
	FILE* imgSaida;

	char *nomeImagem = NULL;
	char *extensao = NULL;
	char c = '\0';
	int i, j;
	int ext = 0;



	int* imgVR;
	int* imgVG;
	int* imgVB;

	int* imgVRF;
	int* imgVGF;
	int* imgVBF;

	int* d_imgVR;
	int* d_imgVG;
	int* d_imgVB;

	int* d_imgVRF;
	int* d_imgVGF;
	int* d_imgVBF;


	int *imgVGray;
	int *imgVGrayFinal;

	int *d_imgVGray;
	int *d_imgVGrayFinal;

	struct timespec clockStart, clockEnd;

	i = 0;
	j = 0;


	printf("Insira o caminho da imagem .ppm ou .pgm: ");
	while(c != '\n')
	{
		c = getchar();
		i++;
		nomeImagem = (char*) realloc(nomeImagem, sizeof(char)*i);
		nomeImagem[i-1] = c;

		if(c == '.')
			ext = 1;

		if(ext == 1)
		{
			j++;
			extensao = (char*) realloc(extensao, sizeof(char)*j);
			extensao[j-1] = c;
		}
	}
	nomeImagem[i-1] = '\0';
	extensao[j-1] = '\0';

	if(!strcmp(extensao, ".ppm"))
	{
		printf("Executando .PPM\n");

		img = fopen(nomeImagem, "r");
		if(img == NULL)
		{
			printf("Erro ao abrir o arquivo %s\n", nomeImagem);
			exit(EXIT_FAILURE);
		}

		lerHeader(&head, img);

		nColunas = head.x;
	    nLinhas = head.y;

	    n = nLinhas*nColunas;

	    //Calcula a quantidade de threads para cada Bloco
	    if(nColunas < MAXTHREADS)
			nTX = nColunas;
		else
			nTX = MAXTHREADS;

		if(nLinhas < MAXTHREADS)
			nTY = nLinhas;
		else
			nTY = MAXTHREADS;

		//Declara a quantidade de Threads por Bloco e o número de Blocos
		dim3 threadsPorBloco(nTX, nTY);
		dim3 numBlocos((nColunas/threadsPorBloco.x) + nColunas%threadsPorBloco.x, (nLinhas/threadsPorBloco.y)+ nLinhas%threadsPorBloco.y);


		size_t size = n*sizeof(int);



		imgVR =(int*) malloc(sizeof(int) * size);
		imgVG =(int*) malloc(sizeof(int) * size);				
		imgVB =(int*) malloc(sizeof(int) * size);

		imgVRF =(int*) malloc(sizeof(int) * size);
		imgVGF =(int*) malloc(sizeof(int) * size);				
		imgVBF =(int*) malloc(sizeof(int) * size);




		lerImagemVetor(&head, img, imgVR, imgVG, imgVB);
	
		imgSaida = fopen("out.ppm", "w");
		if(imgSaida == NULL)
		{
			printf("Erro ao criar arquivo out.ppm\n");
			exit(EXIT_FAILURE);
		}
	
		clock_gettime(CLOCK_MONOTONIC, &clockStart);

		//Uso de recursos do Dispositivo, Contagem do tempo!

		//hipMalloc(&d_imgVR, size);
		//hipMalloc(&d_imgVG, size);
		//hipMalloc(&d_imgVB, size);

		//hipMalloc(&d_imgVRF, size);
		//hipMalloc(&d_imgVGF, size);
		//hipMalloc(&d_imgVBF, size);

		//hipMemcpy(d_imgVR, imgVR, size, hipMemcpyHostToDevice);
		//hipMemcpy(d_imgVG, imgVG, size, hipMemcpyHostToDevice);
		//hipMemcpy(d_imgVB, imgVB, size, hipMemcpyHostToDevice);


		//Processa canal Red

		hipMalloc(&d_imgVR, size);
		hipMalloc(&d_imgVRF, size);
		hipMemcpy(d_imgVR, imgVR, size, hipMemcpyHostToDevice);
		
		filtroGuassiano<<<numBlocos, threadsPorBloco>>>(d_imgVR, d_imgVRF, nLinhas, nColunas, 5);
		hipDeviceSynchronize();

		error = hipGetLastError();
		if(error != hipSuccess)
	    	{
	            printf("Cuda ERROR K1: %s\n", hipGetErrorString(error));
	    	}
		
		hipMemcpy(imgVRF, d_imgVRF, size, hipMemcpyDeviceToHost );
		error = hipGetLastError();
	    	if(error != hipSuccess)
	    	{
	            printf("Cuda ERROR MemCpy1: %s\n", hipGetErrorString(error));
	    	}

		hipFree(d_imgVR);
		hipFree(d_imgVRF);


		//Processa Canal Green

		hipMalloc(&d_imgVG, size);
		hipMalloc(&d_imgVGF, size);
		hipMemcpy(d_imgVG, imgVG, size, hipMemcpyHostToDevice);

		filtroGuassiano<<<numBlocos, threadsPorBloco>>>(d_imgVG, d_imgVGF, nLinhas, nColunas, 5);
		hipDeviceSynchronize();

		error = hipGetLastError();
		if(error != hipSuccess)
	    	{
	            printf("Cuda ERROR K2: %s\n", hipGetErrorString(error));
	    	
		}

		hipMemcpy(imgVGF, d_imgVGF, size, hipMemcpyDeviceToHost );
		error = hipGetLastError();
	    	if(error != hipSuccess)
	    	{
	            printf("Cuda ERROR MemCpy2: %s\n", hipGetErrorString(error));
	    	}

		hipFree(d_imgVG);
		hipFree(d_imgVGF);

		
		//Processa Canal Blue

		hipMalloc(&d_imgVB, size);
		hipMalloc(&d_imgVBF, size);
		hipMemcpy(d_imgVB, imgVB, size, hipMemcpyHostToDevice);
		
		filtroGuassiano<<<numBlocos, threadsPorBloco>>>(d_imgVB, d_imgVBF, nLinhas, nColunas, 5);
		hipDeviceSynchronize();

		error = hipGetLastError();
	    	if(error != hipSuccess)
	    	{
	            printf("Cuda ERROR K3: %s\n", hipGetErrorString(error));
	    	}	

		hipMemcpy(imgVBF, d_imgVBF, size, hipMemcpyDeviceToHost );
	     	error = hipGetLastError();
	    	if(error != hipSuccess)
	    	{
	            printf("Cuda ERROR MemCpy3: %s\n", hipGetErrorString(error));
	    	}

		hipFree(d_imgVB);
		hipFree(d_imgVBF);

		hipDeviceSynchronize();

	   // hipFree(d_imgVR); 
	    //hipFree(d_imgVG); 
	    //hipFree(d_imgVB); 

	    //hipFree(d_imgVRF); 
	    //hipFree(d_imgVGF); 
	    //hipFree(d_imgVBF); 

	    	hipDeviceReset();

		clock_gettime(CLOCK_MONOTONIC, &clockEnd);
		
		printf("Tempo=> %fs\n", ((double)(clockEnd.tv_nsec - clockStart.tv_nsec)/1000000000) + (clockEnd.tv_sec - clockStart.tv_sec));
	

		escreverImagemVetor(imgVR, imgVG, imgVB, &head, imgSaida);

		free(imgVR);
		free(imgVG);
		free(imgVB);


		free(imgVRF);
		free(imgVGF);
		free(imgVBF);

		fclose(imgSaida);
		fclose(img);

	}
	else if(!strcmp(extensao, ".pgm"))
	{
		printf("Executando .PGM\n");

		img = fopen(nomeImagem, "r");
		if(img == NULL)
		{
			printf("Erro ao abrir o arquivo\n");
			exit(EXIT_FAILURE);
		}
	
		lerHeader(&head, img);
	
		nColunas = head.x;
	    nLinhas = head.y;

	    n = nLinhas*nColunas;

	    //Calcula a quantidade de threads para cada Bloco
	    if(nColunas < MAXTHREADS)
			nTX = nColunas;
		else
			nTX = MAXTHREADS;

		if(nLinhas < MAXTHREADS)
			nTY = nLinhas;
		else
			nTY = MAXTHREADS;

		//Declara a quantidade de Threads por Bloco e o número de Blocos
		dim3 threadsPorBloco(nTX, nTY);
		dim3 numBlocos((nColunas/threadsPorBloco.x) + nColunas%threadsPorBloco.x, (nLinhas/threadsPorBloco.y)+ nLinhas%threadsPorBloco.y);


		size_t size = n*sizeof(int);
	
		imgVGray = (int*) malloc(sizeof(int)*size);
		imgVGrayFinal = (int*) malloc(sizeof(int)*size);
	

		lerImagemGrayVetor(&head, img, imgVGray);
	
		imgSaida = fopen("out.pgm", "w");
		if(imgSaida == NULL)
		{
			printf("Erro ao criar arquivo out.ppm\n");
			exit(EXIT_FAILURE);
		}



		clock_gettime(CLOCK_MONOTONIC, &clockStart);

		hipMalloc(&d_imgVGray, size);
		hipMalloc(&d_imgVGrayFinal, size);

		hipMemcpy(d_imgVGray, imgVGray, size, hipMemcpyHostToDevice);

		//printf("n = %d nTx = %d nTy = %d\n nBx = %d nBy = %d\n", n, threadsPorBloco.x, threadsPorBloco.y, numBlocos.x, numBlocos.y);

		filtroGuassiano<<<numBlocos, threadsPorBloco>>>(d_imgVGray, d_imgVGrayFinal, nLinhas, nColunas, 5);

		hipDeviceSynchronize();

		error = hipGetLastError();
	    if(error != hipSuccess)
	    {
	            printf("Cuda ERROR 1: %s\n", hipGetErrorString(error));
	    }

		hipMemcpy(imgVGrayFinal, d_imgVGrayFinal, size, hipMemcpyDeviceToHost );

		hipDeviceSynchronize();

	    error = hipGetLastError();
	    if(error != hipSuccess)
	    {
	            printf("Cuda ERROR 2: %s\n", hipGetErrorString(error));
	    }


	    hipFree(d_imgVGray);
	    hipFree(d_imgVGrayFinal);

	    hipDeviceReset();

		clock_gettime(CLOCK_MONOTONIC, &clockEnd);

		printf("Tempo=> %fs\n", ((double)(clockEnd.tv_nsec - clockStart.tv_nsec)/1000000000) + (clockEnd.tv_sec - clockStart.tv_sec));

		escreverImagemGrayVetor(imgVGray, &head, imgSaida);

		free(imgVGray);
		free(imgVGrayFinal);

		fclose(imgSaida);
		fclose(img);

	}
	else
	{
		printf("Extensao nao suportada\n");
	}

    return 0;
}
